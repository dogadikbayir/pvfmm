
#include <hip/hip_runtime.h>
#include "stdint.h"

#define DEFAULT_NUM_THREAD 256

/* Case: double */
__global__ void in_perm_k (
  uintptr_t *precomp_data,
  uintptr_t *input_perm,
  uintptr_t *input_data,
  uintptr_t *buff_in,
  size_t interac_indx,
  size_t M_dim0,
  size_t vec_cnt )
{
  /* 1-dim thread Id. */
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  /* Convert to ptr. */
  int *perm = (int*) (precomp_data[0] + input_perm[(interac_indx + tid)*4 + 0]);
  double *scal = (double*) (precomp_data[0] + input_perm[(interac_indx + tid)*4 + 1]);
  double *v_in = (double*) (input_data[0] + input_perm[(interac_indx + tid)*4 + 3]);
  double *v_out = (double*) (buff_in + input_perm[(interac_indx + tid)*4 + 2]);

  if (tid < vec_cnt) {
    /* PRAM Model: assuming as many threads as we need. */
    for (int j = 0; j < M_dim0; j++) v_out[j] = v_in[perm[j]]*scal[j];
  }
}

__global__ void out_perm_k (
  uintptr_t *scaling,
  uintptr_t *precomp_data,
  uintptr_t *output_perm,
  uintptr_t *output_data,
  uintptr_t *buff_out,
  size_t interac_indx,
  size_t M_dim1,
  size_t vec_cnt )
{
  /* 1-dim thread Id. */
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  /* Specifing range. */
  int a = tid;
  int b = tid + 1;

  if (tid > 0 && a < vec_cnt) { // Find 'a' independent of other threads.
    size_t out_ptr = output_perm[(interac_indx + a)*4 + 3];
    if (tid > 0) while(a < vec_cnt && out_ptr == output_perm[(interac_indx+a)*4 + 3]) a++;
  }
  if (tid < vec_cnt - 1 && b < vec_cnt) { // Find 'b' independent of other threads.
    size_t out_ptr = output_perm[(interac_indx + b)*4 + 3];
    if (tid < vec_cnt-1) while(b < vec_cnt && out_ptr == output_perm[(interac_indx+b)*4 + 3]) b++;
  }

  if (tid < vec_cnt) {
    /* PRAM Model: assuming as many threads as we need. */
    for(int i = a; i < b; i++) { // Compute permutations.
      double scaling_factor = scaling[interac_indx + i];
      int *perm = (int*) (precomp_data[0] + output_perm[(interac_indx + i)*4 + 0]);
      double *scal = (double*) (precomp_data[0] + output_perm[(interac_indx + i)*4 + 1]);
      double *v_in = (double*) (buff_out + output_perm[(interac_indx + i)*4 + 3]);
      double *v_out = (double*) (output_data[0] + output_perm[(interac_indx + i)*4 + 2]);
      for (int j = 0; j < M_dim1; j++) v_out[j] += v_in[perm[j]]*scal[j]*scaling_factor;
    }
  }
}

void in_perm_d (
  uintptr_t *precomp_data,
  uintptr_t *input_perm,
  uintptr_t *input_data,
  uintptr_t *buff_in,
  size_t interac_indx,
  size_t M_dim0,
  size_t vec_cnt,
  hipStream_t *stream )
{
  int n_thread, n_block;
  n_thread = DEFAULT_NUM_THREAD;
  n_block = vec_cnt/n_thread;
  in_perm_k<<<n_thread, n_block, 0, *stream>>>(precomp_data, input_perm, input_data, buff_in, interac_indx, M_dim0, vec_cnt);
}

void out_perm_d (
  uintptr_t *scaling,
  uintptr_t *precomp_data,
  uintptr_t *output_perm,
  uintptr_t *output_data,
  uintptr_t *buff_out,
  size_t interac_indx,
  size_t M_dim0,
  size_t vec_cnt,
  hipStream_t *stream )
{
  int n_thread, n_block;
  n_thread = DEFAULT_NUM_THREAD;
  n_block = vec_cnt/n_thread;
  out_perm_k<<<n_thread, n_block, 0, *stream>>>(scaling, precomp_data, output_perm, output_data, buff_out, interac_indx, M_dim0, vec_cnt);
}
